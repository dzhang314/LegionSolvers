#include "CSRMatrixTasks.hpp"

#include "CuSPARSEHelpers.hpp"
#include "CudaLibs.hpp"
#include "LegionUtilities.hpp" // for AffineReader, AffineWriter, ...
#include "LibraryOptions.hpp"  // for LEGION_SOLVERS_USE_*

using namespace LegionSolvers;

template <
    typename ENTRY_T,
    int KERNEL_DIM,
    int DOMAIN_DIM,
    int RANGE_DIM,
    typename KERNEL_COORD_T,
    typename DOMAIN_COORD_T,
    typename RANGE_COORD_T>
void CSRMatvecTask<
    ENTRY_T,
    KERNEL_DIM,
    DOMAIN_DIM,
    RANGE_DIM,
    KERNEL_COORD_T,
    DOMAIN_COORD_T,
    RANGE_COORD_T>::
    cuda_task_body(
        const Legion::Task *task,
        const std::vector<Legion::PhysicalRegion> &regions,
        Legion::Context ctx,
        Legion::Runtime *rt
    ) {

    assert(regions.size() == 4);
    const auto &output_vec = regions[0];
    const auto &csr_matrix = regions[1];
    const auto &aux_region = regions[2];
    const auto &input_vec = regions[3];

    assert(task->regions.size() == 4);
    const auto &output_req = task->regions[0];
    const auto &matrix_req = task->regions[1];
    const auto &aux_req = task->regions[2];
    const auto &input_req = task->regions[3];

    assert(output_req.privilege_fields.size() == 1);
    const Legion::FieldID output_fid = *output_req.privilege_fields.begin();

    assert(matrix_req.privilege_fields.size() == 2);
    assert(task->arglen == 2 * sizeof(Legion::FieldID));
    const Legion::FieldID *argptr =
        reinterpret_cast<const Legion::FieldID *>(task->args);
    const Legion::FieldID fid_col = argptr[0];
    const Legion::FieldID fid_entry = argptr[1];

    assert(aux_req.privilege_fields.size() == 1);
    const Legion::FieldID fid_rowptr = *aux_req.privilege_fields.begin();

    assert(input_req.privilege_fields.size() == 1);
    const Legion::FieldID input_fid = *input_req.privilege_fields.begin();

    const AffineReader<
        Legion::Point<DOMAIN_DIM, DOMAIN_COORD_T>,
        KERNEL_DIM,
        KERNEL_COORD_T>
        col_reader{csr_matrix, fid_col};

    const AffineReader<ENTRY_T, KERNEL_DIM, KERNEL_COORD_T> entry_reader{
        csr_matrix, fid_entry};

    const AffineReader<
        Legion::Rect<KERNEL_DIM, KERNEL_COORD_T>,
        RANGE_DIM,
        RANGE_COORD_T>
        rowptr_reader{aux_region, fid_rowptr};

    const AffineReader<ENTRY_T, DOMAIN_DIM, DOMAIN_COORD_T> input_reader{
        input_vec, input_fid};

    const AffineSumAccessor<ENTRY_T, RANGE_DIM, RANGE_COORD_T> output_writer{
        output_vec, output_fid, LEGION_REDOP_SUM<ENTRY_T>};

    auto stream = get_cached_stream();
    auto handle = get_cusparse();
    CHECK_CUSPARSE(hipsparseSetStream(handle, stream));

    auto output_bounds = output_vec.get_bounds<RANGE_DIM, RANGE_COORD_T>();
    assert(output_bounds.dense());
    auto rows = output_bounds.bounds.volume();

    auto input_bounds = input_vec.get_bounds<DOMAIN_DIM, DOMAIN_COORD_T>();
    // The number of columns in this slice of the COO matrix is at most
    // the upper domain of the input vector, since the kernel and domain
    // are related by an image.
    static_assert(DOMAIN_DIM == 1);
    auto cols = input_bounds.bounds.hi[0] + 1;

    auto cusparse_csr = makeCuSparseCSR<
        ENTRY_T,
        KERNEL_DIM,
        DOMAIN_DIM,
        RANGE_DIM,
        KERNEL_COORD_T,
        DOMAIN_COORD_T,
        RANGE_COORD_T>(
        stream,
        rows,
        cols,
        aux_region.get_bounds<KERNEL_DIM, KERNEL_COORD_T>(),
        rowptr_reader,
        csr_matrix.get_bounds<KERNEL_DIM, KERNEL_COORD_T>(),
        col_reader,
        entry_reader
    );
    // There is an image relationship between col->input, so input should
    // be offset to the base of the image rather used directly.
    auto cusparse_input =
        makeShiftedCuSparseDnVec<ENTRY_T, decltype(input_reader)>(
            input_bounds, cols, input_reader
        );
    // There is no such relationship between output and row, so the
    // vector can be used directly.
    auto cusparse_output = makeCuSparseDnVec<ENTRY_T, decltype(output_writer)>(
        output_bounds, output_writer
    );

    ENTRY_T alpha = static_cast<ENTRY_T>(1.0);
    ENTRY_T beta = static_cast<ENTRY_T>(0.0);
    size_t bufSize = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,
        cusparse_csr,
        cusparse_input,
        &beta,
        cusparse_output,
        cusparseDataType<ENTRY_T>(),
        HIPSPARSE_MV_ALG_DEFAULT,
        &bufSize
    ));
    void *workspace = nullptr;
    if (bufSize > 0) {
        Legion::DeferredBuffer<char, 1> buf(
            {0, bufSize - 1}, Legion::Memory::GPU_FB_MEM
        );
        workspace = buf.ptr(0);
    }
    CHECK_CUSPARSE(hipsparseSpMV(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,
        cusparse_csr,
        cusparse_input,
        &beta,
        cusparse_output,
        cusparseDataType<ENTRY_T>(),
        HIPSPARSE_MV_ALG_DEFAULT,
        workspace
    ));

    // Clean up after ourselves.
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_input));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(cusparse_output));
    CHECK_CUSPARSE(hipsparseDestroySpMat(cusparse_csr));
}

template <
    typename ENTRY_T,
    int KERNEL_DIM,
    int DOMAIN_DIM,
    int RANGE_DIM,
    typename KERNEL_COORD_T,
    typename DOMAIN_COORD_T,
    typename RANGE_COORD_T>
void CSRRmatvecTask<
    ENTRY_T,
    KERNEL_DIM,
    DOMAIN_DIM,
    RANGE_DIM,
    KERNEL_COORD_T,
    DOMAIN_COORD_T,
    RANGE_COORD_T>::
    cuda_task_body(
        const Legion::Task *task,
        const std::vector<Legion::PhysicalRegion> &regions,
        Legion::Context ctx,
        Legion::Runtime *rt
    ) {
    assert(false);
}

template void LegionSolvers::CSRMatvecTask<
    float,
    1,
    1,
    1,
    Legion::coord_t,
    Legion::coord_t,
    Legion::coord_t>::
    cuda_task_body(
        const Legion::Task *task,
        const std::vector<Legion::PhysicalRegion> &regions,
        Legion::Context ctx,
        Legion::Runtime *rt
    );
template void LegionSolvers::CSRMatvecTask<
    double,
    1,
    1,
    1,
    Legion::coord_t,
    Legion::coord_t,
    Legion::coord_t>::
    cuda_task_body(
        const Legion::Task *task,
        const std::vector<Legion::PhysicalRegion> &regions,
        Legion::Context ctx,
        Legion::Runtime *rt
    );
